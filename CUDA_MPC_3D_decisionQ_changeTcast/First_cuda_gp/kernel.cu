#include "hip/hip_runtime.h"
//��MPCģ���н�h��ΪQ,���ٱ��ֲ���1.2m/min,�ı佽ע�¶�
#include "hip/hip_runtime.h"
#include ""
#include <cuda_occupancy.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <fstream>
#include "book.h"
#include "gridcheck.h"
#include <string>
using namespace std;
# define Section 12  // number of cooling sections
# define CoolSection 8
# define MoldSection 4
# define StaticIter 50
# define M 3
# define N M+2*CoolSection
# define TestIter 1000
# define limit 1001//limit>=tnpts/num_iter

float ccml[Section + 1] = { 0.0,0.2,0.4,0.6,0.8,1.0925,2.27,4.29,5.831,9.6065,13.6090,19.87014,28.599 }; // The cooling sections
float H_Init[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
//float H_Init[Section] = { 1400,1200,1000,800,1200,750,400,400,350,300,250,150 };
//float H_Init[Section] = { 1500,1300,1100,900,1300,850,500,500,450,400,350,250 };
//float H_Init_Temp[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
float H_Init_Temp[Section] = { 0 };
float H_Init_Final[Section] = { 1380 };
float Q_air[CoolSection] = { 200,1500,850,650,1000,850,400,480 };
float Taim[CoolSection] = { 966.149841, 925.864746, 952.322083, 932.175537, 914.607117, 890.494263, 870.804443, 890.595825 };
float delta_z[Section] = {2.7,2.7,1.8,1.8,1.8,1.8,1.8,0.9};
float *Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T, float num);
float *calculateThickness(float *T_result, int nx, int ny, int nz, float dy, float *ccml, float Ts, float thick);
hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy);
__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd);
__device__ float Boundary_Condition(int j, float dx, float *ccml_zone, float *H_Init);
float *relationshiphandQ(float *h_Init, float* Q_air);
float stop_criterion();
float update_c(float[], float c0,int iter);
void update_lamda(float[],int iter,float[]);
float alfa[limit] = { 1.0 };
float g[N] = { 0 };
float testArray[TestIter] = { 0 };

__global__ void addKernel(float *T_New, float *T_Last, float *ccml, float *H_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, bool disout,float Vcast,float T_casting)
{
	int i = threadIdx.x;
	int m = threadIdx.y;
	int j = blockIdx.x;
	int idx = j * nx * nz + m * nx + i;
	int ND = nx * nz;
	int D = nx;

	float pho, Ce, lamd; // physical parameters pho represents desity, Ce is specific heat and lamd is thermal conductivity
	float a, T_Up, T_Down, T_Right, T_Left, T_Forw, T_Back, h = 100.0, Tw = 30.0; //Vcast = -0.02, T_Cast = 1558.0

	if (disout) {
		Physicial_Parameters(T_Last[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_New[idx] = T_casting;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = (a*tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ (a*tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}

	else
	{
		Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_Last[idx] = T_casting;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}
}
int main()
{
	const int nx = 21, ny = 3000, nz = 21;   // nx is the number of grid in x direction, ny is the number of grid in y direction.
	int num_blocks = 1, num_threadsx = 1, num_threadsy = 1;// num_threadsz = 1; // block number(1D)  thread number in x and y dimension(2D)
	int tnpts = 10001;  // time step
	float  Lx = 0.25, Ly = 28.599, Lz = 0.25, t_final = 2000.0, dx, dy, dz, tao;  //T_Cast = 1558.0, T_Cast is the casting temperature Lx and Ly is the thick and length of steel billets
	
	float *T_Init;
	num_threadsx = nx;
	num_threadsy = nz;
	num_blocks = ny;

	T_Init = (float*)calloc(nx*ny*nz,sizeof(float));  // Initial condition

	//for (int m = 0; m < nz; m++)
	//	for (int j = 0; j < ny; j++)
	//       for (int i = 0; i < nx; i++)
	//		   T_Init[nx * ny * m + j * nx + i] = T_Cast;  // give the initial condition

	//��ȡtxt�ļ�
	ifstream in("F:\\Temperature3DGPU_shared_memmory_Static.txt");
	if (!in)
	{
		cerr << "open the filename failed!" << endl;
		return 1;
	}
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			for (int m = 0; m < nz; m++)
				in >> T_Init[nx * nz * j + i * nz + m];
		}
	}
	in.close();

	ofstream fout;
	fout.open("F:\\data_zf\\testTinit.txt");
	if (!fout)
		cout << "testTinit is not open" << endl;
	else
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				for (int m = 0; m < nz; m++)
					fout << T_Init[nx * nz * j + i * nz + m] << ", ";
				fout << endl;
			}
			fout << endl;
		}
	}
	fout.close();


	dx = Lx / (nx - 1);            // the grid size x
	dy = Ly / (ny - 1);            // the grid size y
	dz = Lz / (nz - 1);            // the grid size z
	tao = t_final / (tnpts - 1);   // the time step size
	//gridcheck(dx, dy, tao);

	//cout << "Casting Temperature " << T_Cast << endl;
	cout << "The length of steel billets(m) " << Ly << endl;
	cout << "The width of steel billets(m) " << Lz << endl;
	cout << "The thick of steel billets(m) " << Lx << endl;
	cout << "dx(m) " << dx << ", ";
	cout << "dy(m) " << dy << ", ";
	cout << "dz(m) " << dz << ", ";
	cout << "tao(s) " << tao << ", ";
	cout << "simulation time(s) " << t_final << endl;

	//clock_t timestart = clock();

	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	hipError_t cudaStatus = addWithCuda(T_Init, dx, dy, dz, tao, nx, ny, nz, tnpts, num_blocks, num_threadsx, num_threadsy);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float elapsetime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsetime, start, stop));
	cout << "running time =" << (elapsetime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	/*clock_t timeend = clock();

	cout << "running time = " << (timeend - timestart);*/

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy)
{
	float *dev_T_New, *dev_T_Last, *dev_ccml, *dev_H_Init; // the point on GPU
	float *T_Result, *Delta_H_Init, *T_HoldLast,*ThickAll, **Mean_TSurfaceElement, **Mean_TSurfaceElementOne;
	float *Point_TSurfaceElement, *Point_TSurfaceElementOne, **Mean_TCenterElement, **Mean_TCenterElementOne;
	float **JacobianMatrix, *JacobianG0, *JacobianG1, *JacobianG2, *TZ_gradient,*partionQ;
	float **JacobinTZgradient, **TZ_gradientElement, **TZ_gradientElementOne;
	float dh = 10.0,dQ=1.0, arf1, arf2, step = -0.0001,T_bmax=1100,Ts=1462,Tl= 1518.0,Tu=-100,Td=200;
	float Vcast = -0.02, T_casting = 1558.0;
	const int Num_Iter = 10, PrintLabel = 0;// The result can be obtained by every Num_Iter time step
	volatile bool dstOut = true;

	//Լ������
	float c[limit] = {10};
	float norm_g[limit] = { 0 };
	float eps = 0.0001,c0 = 10;	
	float lamda[limit][N] = { 1 };
	float gtest[limit][N] = { 0 };
	float htest[limit][Section] = { 0 };
	float fitness[limit] = { 0 };
	float gfitness[limit] = { 0 };

	T_Result = (float *)calloc(nx * ny * nz, sizeof(float)); // The temperature of steel billets
	Delta_H_Init = (float*)calloc(CoolSection, sizeof(float));
	T_HoldLast = (float*)calloc(nz * ny * nx, sizeof(float));
	Point_TSurfaceElement = (float*)calloc(CoolSection, sizeof(float));
	Point_TSurfaceElementOne = (float*)calloc(CoolSection, sizeof(float));
	JacobianG0 = (float*)calloc(CoolSection, sizeof(float));
	JacobianG1 = (float*)calloc(CoolSection, sizeof(float));
	JacobianG2 = (float*)calloc(CoolSection, sizeof(float));
	TZ_gradient = (float*)calloc(CoolSection, sizeof(float));
	partionQ = (float*)calloc(CoolSection, sizeof(float));
	ThickAll = (float*)calloc(Section, sizeof(float));

	JacobianMatrix = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		JacobianMatrix[i] = (float*)calloc(CoolSection, sizeof(float));
	
	Mean_TSurfaceElement = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)	
		Mean_TSurfaceElement[i] = (float*)calloc(CoolSection, sizeof(float));	

	Mean_TSurfaceElementOne = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TSurfaceElementOne[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TCenterElement = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TCenterElement[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TCenterElementOne = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TCenterElementOne[i] = (float*)calloc(CoolSection, sizeof(float));

	JacobinTZgradient = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++) 
		JacobinTZgradient[i] = (float*)calloc(CoolSection, sizeof(float));
	
	TZ_gradientElement = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		TZ_gradientElement[i] = (float*)calloc(CoolSection, sizeof(float));

	TZ_gradientElementOne = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		TZ_gradientElementOne[i] = (float*)calloc(CoolSection, sizeof(float));

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ccml, (Section + 1) * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init, Section * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_T_Last, T_Init, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_ccml, ccml, (Section + 1) * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init, Section * sizeof(float), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(num_threadsx, num_threadsy);
	float SurfaceError[TestIter / 10+1][CoolSection];
	for (int t = 0; t < TestIter*10+1; t++)
	{			
		if ((t / Num_Iter) > (8*StaticIter))
			T_casting = 1658;
		if (t % Num_Iter == 0)
		{
			int iter = t / Num_Iter;
			HANDLE_ERROR(hipMemcpy(T_HoldLast, dev_T_Last, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
			for (int m = 0; m < CoolSection + 1; m++)
			{			
				if (m == CoolSection)
				{
					for (int temp = 0; temp < Section; temp++) {
						H_Init_Temp[temp] = H_Init[temp];
					}
						
					HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));
					for (int PNum = 0; PNum < Num_Iter; PNum++)
					{
						addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut,Vcast,T_casting);
						dstOut = !dstOut;
					}

					HANDLE_ERROR(hipMemcpy(T_Result, dev_T_New, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
					float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result,0);  // calculation the mean surface temperature of steel billets in every cooling sections
					float* Mean_TPoint = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result, 8.0 / 250 * nx);//һ������¶�
					float Point_TSurface = Mean_TPoint[MoldSection];
					float *Mean_TCenter = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result,nx/2);//�����¶ȣ�Ϊɶ�������أ�
					for (int temp = 0; temp < CoolSection; temp++) {
						
						Point_TSurfaceElementOne[temp] = Point_TSurface;
						if (iter >= StaticIter) 
						{
							if (temp < CoolSection - 1)
								TZ_gradient[temp] = (Mean_TSurface[temp + 1 + MoldSection] - Mean_TSurface[temp + MoldSection]) / delta_z[temp];
							else
								//TZ_gradient[temp] = -(T_Result[nx*nz*(ny - 1) + 0 * nz + (int)(nx - 1)] - Mean_TSurface[temp + MoldSection]) / delta_z[temp];	
								TZ_gradient[temp] = 100;
							//printf("TZ_gradient=%f  ", TZ_gradient[temp]);
						}
						for (int column = 0; column < CoolSection; column++) 
						{
							Mean_TSurfaceElementOne[temp][column] = Mean_TSurface[column + MoldSection];
							Mean_TCenterElementOne[temp][column] = Mean_TCenter[column + MoldSection];
							TZ_gradientElementOne[temp][column] = TZ_gradient[column + MoldSection];
						}
					}
					//printf("\n");
				}

				else
				{				

					for (int temp = 0; temp < Section; temp++) 						
						H_Init_Temp[temp] = H_Init[temp];			
					//printf("  h=%f", H_Init_Temp[m]);
					H_Init_Temp[m + MoldSection] = H_Init[m + MoldSection] + dh;
					HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));

					for (int PNum = 0; PNum < Num_Iter; PNum++)//Ԥ��ʱ�γ���
					{
						addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut,Vcast,T_casting);
						dstOut = !dstOut;
					}

					HANDLE_ERROR(hipMemcpy(T_Result, dev_T_New, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
					float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result,0); // calculation the mean surface temperature of steel billets in every cooling sections
					float* Mean_TPoint = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result, 8.0 / 250 * nx);
					Point_TSurfaceElement[m] = Mean_TPoint[m];
					float *Mean_TCenter = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result,nx/2);					
					if (iter >= 2 * StaticIter) 
					{
						if (m < CoolSection - 1)
							TZ_gradient[m] = (Mean_TSurface[m + 1 + MoldSection] - Mean_TSurface[m + MoldSection]) / delta_z[m];
						else
							//TZ_gradient[m] = -(T_Result[nx*nz*(ny - 1) + 0 * nz + (int)(nx - 1)] - Mean_TSurface[m + MoldSection]) / delta_z[m];
							TZ_gradient[m] = 150;
					}				
					for (int column = 0; column < CoolSection; column++)
					{						
						Mean_TSurfaceElement[m][column] = Mean_TSurface[column + MoldSection];//�Ͷ�������Ӧ
						Mean_TCenterElement[m][column] = Mean_TCenter[column + MoldSection];
						TZ_gradientElement[m][column] = TZ_gradient[column + MoldSection];
					}					
					/*if (iter >= StaticIter) {
						g[0] = Mean_TSurface[MoldSection] - T_bmax;
						g[1] = Mean_TPoint[MoldSection]-Ts;
						g[2] = Mean_TCenter[MoldSection + CoolSection - 2] - Tl;
					}*/
				}
				/*for (int i = 0; i < CoolSection; i++) {
					printf("TZ_gradient=%f  ",TZ_gradient[i]);
				}
				printf("\n");*/
				
				//���Լ��,��һ�ζ�������ƽ���¶� 
				if (iter >= StaticIter)
				{
					g[0] = Mean_TSurfaceElement[0][0] - T_bmax;
					g[2] = Mean_TCenterElement[MoldSection + 1][MoldSection + 1] - Tl;
					g[1] = Point_TSurfaceElement[MoldSection] - Ts;
					/*printf("g[0]=%f\n", g[0]);
					printf("g[1]=%f\n", g[1]);
					printf("g[2]=%f\n", g[2]);*/
					for (int i = M; i < N; i++) {
						if (i < M + CoolSection)
							g[i] = Tu - TZ_gradient[i-M];
						else
							g[i] = TZ_gradient[i - M - CoolSection]-Td;						
					}				
				}
				for (int temp = 0; temp < M; temp++)
					gfitness[iter] += lamda[iter][temp] * g[temp];
				for (int i = 0; i < N; i++)
				{
					if (iter < StaticIter)
						gtest[iter][i] = 0;
					else
						gtest[iter][i] = g[i];
				}			
				HANDLE_ERROR(hipMemcpy(dev_T_Last, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));

			}

			printf("iter=%d\n", iter);
			printf("g[0]=%f\n", g[0]);
			//�ж��Ƿ�����ѭ��
			if (iter >= StaticIter)
			{
				norm_g[iter] = stop_criterion();
				norm_g[0] = norm_g[StaticIter];
			}
			if (norm_g[iter - 1]<eps&&iter>2*StaticIter)//����ֹͣ׼��
				break;

			//���³���			
			c[iter] = update_c(norm_g, c0, iter-StaticIter);
			printf("c=%f\n", c[iter]);

			if (iter <= StaticIter)
				for (int j = 0; j < N; j++) 
					lamda[iter][j] = 1;
			/*for (int j = 0; j < N; j++) {
				printf("lamda[i]=%f\n", lamda[iter][j]);
				printf("g[i]=%f\n", g[j]);
			}*/

			for (int j = 0; j < Section; j++) {
				htest[iter][j] = H_Init_Temp[j];
			}
			//Ŀ�꺯���ݶ�
			for (int row = 0; row < CoolSection; row++)
			{
				for (int column = 0; column < CoolSection; column++)
				{
					JacobianMatrix[row][column] = (Mean_TSurfaceElement[row][column] - Mean_TSurfaceElementOne[row][column]) / dh;	//���ϵ���1
					JacobinTZgradient[row][column] = (TZ_gradientElement[row][column] - TZ_gradientElementOne[row][column]) / dh;
					if (row == 0)
					{
						if (iter > StaticIter)
						{
							JacobianG0[column] = (Mean_TSurfaceElement[0][column] - Mean_TSurfaceElementOne[0][column]) / dh;
							JacobianG1[column] = (Point_TSurfaceElement[column]- Point_TSurfaceElementOne[column]) / dh;
							JacobianG2[column] = (Mean_TCenterElement[4][column] - Mean_TCenterElementOne[4][column]) / dh;
						}
					}
				}
			}				


			for (int temp = 0; temp < CoolSection; temp++) {
				Delta_H_Init[temp] = 0.0;
				for (int column = 0; column < CoolSection; column++)
				{
					Delta_H_Init[temp] += (Mean_TSurfaceElementOne[temp][column] - Taim[column]) * JacobianMatrix[temp][column];//���ϵ���2
					if (iter > StaticIter)
					{
						Delta_H_Init[temp] += lamda[iter][temp + M] * JacobinTZgradient[temp][column];
						Delta_H_Init[temp] +=(-1)* lamda[iter][temp + M + CoolSection] * JacobinTZgradient[temp][column];
					}					
					
				}
				Delta_H_Init[temp] += H_Init[temp] - H_Init_Final[temp];//���ӵ�h����������
				gfitness[iter] += lamda[iter][temp + M] * g[temp + M];
				gfitness[iter] += lamda[iter][temp + M + CoolSection] * g[temp + M + CoolSection];
				fitness[iter]+= pow(H_Init[temp] - H_Init_Final[temp],2);
				gfitness[iter] += pow(H_Init[temp] - H_Init_Final[temp], 2);

				
				if (iter > StaticIter)
				{
					Delta_H_Init[temp] += lamda[iter][0] * JacobianG0[temp];//������ʽ�ĵ�������1
					Delta_H_Init[temp] += lamda[iter][1] * JacobianG1[temp];//������ʽ�ĵ�������2
					Delta_H_Init[temp] += lamda[iter][2] * JacobianG2[temp];//������ʽ�ĵ�������3

				}
				//printf("  Delta_H_Init=%f\n", Delta_H_Init[temp]);
			}

			printf("\n");



			//��������Ŀ�꺯���ı�
			arf1 = 0.0, arf2 = 0.0;
			for (int temp = 0; temp < CoolSection; temp++)
			{
				for (int column = 0; column < CoolSection; column++)
				{
					arf1 += ((Mean_TSurfaceElementOne[0][temp] - Taim[temp]) * JacobianMatrix[temp][column]) * Delta_H_Init[column];
					if (iter > StaticIter)
					{
						arf1 += (lamda[iter][temp + M] * JacobinTZgradient[temp][column] * Delta_H_Init[column]);
						arf1 += (lamda[iter][temp + M + CoolSection] * (-1)*JacobinTZgradient[temp][column] * Delta_H_Init[column]);
					}
					arf2 += JacobianMatrix[temp][column] * Delta_H_Init[column] * JacobianMatrix[temp][column] * Delta_H_Init[column];
				}
				//arf1 += (H_Init[temp] - H_Init_Final[temp])*Delta_H_Init[temp];
				//if (iter > StaticIter)
				{
					arf1 += lamda[iter][0] * JacobianG0[temp] * Delta_H_Init[temp];//������ʽ����1
					arf1 += lamda[iter][1] * JacobianG1[temp] * Delta_H_Init[temp];//������ʽ����2
					arf1 += lamda[iter][2] * JacobianG2[temp] * Delta_H_Init[temp];//������ʽ����2
				}
			}
			step = -arf1 / ((arf2)+0.001);//������ʽ����Ŀ�꺯���ı�,ΪɶҪ��0.001��			
			testArray[iter] = step;
			printf("step=%f\n", step);

			//��������
			for (int temp = 0; temp < CoolSection; temp++)
			{
				H_Init_Final[temp] = H_Init[temp];
				H_Init[temp + MoldSection] += step *Delta_H_Init[temp];			
				//printf("  h=%f", H_Init[temp + MoldSection]);
			}			
			float *Q_water = relationshiphandQ(H_Init, Q_air);
			/*for (int temp = 0; temp < CoolSection; temp++) {
				printf("  Q_water=%f", Q_water[temp]*1000/60);
			}*/
			if (iter >= StaticIter)
			{
				for (int j = 0; j < N; j++)
				{
					lamda[iter + 1][j] = lamda[iter][j] + c[iter] * g[j];//lamda�ĸ���
					if (lamda[iter + 1][j] < 0)
						lamda[iter + 1][j] = 0;//��֤ϵ��lamda�ķǸ���
					if (lamda[iter + 1][j] > 100)
						lamda[iter][j] /= lamda[iter][j];
				}
			}
		}
		
			//ʵ��ģ����������
		for (int temp = 0; temp < Section; temp++)
		{
			H_Init_Temp[temp] = H_Init[temp];

		}
		     
			HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));
			addKernel << <num_blocks, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut,Vcast,T_casting);
			dstOut = !dstOut;
			HANDLE_ERROR(hipMemcpy(T_Result, dev_T_Last, nx * ny * nz* sizeof(float), hipMemcpyDeviceToHost));
			float* Mean_TSurface = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result, 0);  // calculation the mean surface temperature of steel billets in every cooling sections
			
			for (int temp = 0; temp < CoolSection; temp++) {
				fitness[t / Num_Iter] += pow((Mean_TSurface[temp + MoldSection] - Taim[temp]), 2);
				gfitness[t / Num_Iter] += pow((Mean_TSurface[temp + MoldSection] - Taim[temp]), 2);
			}
			
			if (t % (10 * Num_Iter) == 0)
			{				
				//�ᾧ����Լ��
                /*int thickness = 0;
				for (; thickness < nx / 2; thickness++) {
					float *Mean_Thickness = Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result, thickness);
					if (Mean_Thickness[MoldSection] > Ts)
						break;					
				}*/
				//printf("thickness=%d\n", thickness);
				

				ThickAll = calculateThickness(T_Result, nx, ny, nz, dy, ccml, Ts, 250);//���Ϊ0.25m				
				cout << endl<<"  ThickAll= " << endl;
				for (int temp = 0; temp < CoolSection; temp++)
					cout << ThickAll[temp + MoldSection] << ", ";
				

				cout << "  time_step = " << t << ",  " << "simulation time = " << t * tao;
				cout << endl << "TSurface = " << endl;
				for (int temp = 0; temp < CoolSection; temp++)
					cout << Mean_TSurface[temp + MoldSection] << ", ";

				cout << endl << "TSurface - Taim = " << endl;
				for (int temp = 0; temp < CoolSection; temp++)
				{
					cout << (Mean_TSurface[temp + MoldSection] - Taim[temp]) << ", ";
					SurfaceError[t / (10 * Num_Iter)][temp] = (Mean_TSurface[temp + MoldSection] - Taim[temp]);
				}
				cout << endl;
			}	
	}
	    
	ofstream fout;
		fout.open("F:\\data_zf\\changeTcastGPUMPC3D2block3threads.txt");
		if (!fout)
			cout << "changeTcastGPUMPC3D2block3threads is not open" << endl;
		else
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					for (int m = 0; m < nz; m++)
						fout << T_Result[nx * nz * j + i * nz + m] << ", ";
					fout << endl;
				}
				fout << endl;
			}
		}		
		fout.close();

		fout.open("F:\\data_zf\\changeTcastSurfaceGPUMPC3D2block3threads.txt");
		if (!fout)
			cout << "changeTcastSurfaceGPUMPC3D2block3threads is not open" << endl;
		else 
		{
			for (int j = 0; j < ny; j++)
			{
				fout << T_Result[nx * nz * j + 0 * nz + int((nx - 1) / 2)] << ", ";
			
				fout << endl;
			}
		}		
		fout.close();
	    
		fout.open("F:\\data_zf\\changeTcastCenterGPUMPC3D2block3threads.txt");
		if (!fout)
			cout << "changeTcastCenterGPUMPC3D2block3threads is not open" << endl;
		else
		{
			for (int j = 0; j < ny; j++)
			{
				fout << T_Result[nx * nz * j + int((nx - 1) / 2) * nz + int((nx - 1) / 2)] << ", ";

				fout << endl;
			}
		}
		fout.close();

		fout.open("F:\\data_zf\\changeTcastSurfaceErrorGPUMPC3D2block3threads.txt");
		if (!fout)
			cout << "changeTcastSurfaceErrorGPUMPC3D2block3threads is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter / 10+1; i++)
			{
				for (int j = 0; j < CoolSection; j++)
					fout << SurfaceError[i][j] << ",";
				fout << endl;
			}
		}					
		fout.close();

		fout.open("F:\\data_zf\\changeTcastThicknessGPUMPC3D2block3threads.txt");
		if (!fout)
			cout << "changeTcastThicknessGPUMPC3D2block3threads is not open" << endl;
		else
		{
			for (int i = 0; i < CoolSection; i++) {
				fout << ThickAll[i + MoldSection] << ",";
				fout << endl;
			}
			
		}
		fout.close();

		fout.open("F:\\data_zf\\changeTcastlamda.txt");
		if (!fout)
			cout << "changeTcastlamda is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				for (int j = 0; j < N; j++)
					fout << lamda[i][j] << ",";
				fout << endl;			
			}
		}	
		fout.close();

		fout.open("F:\\data_zf\\changeTcastgtest.txt");
		if (!fout)
			cout << "changeTcastgtest is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				for (int j = 0; j < N; j++)
					fout << gtest[i][j] << ",";
				fout << endl;
			}
		}		
		fout.close();

		fout.open("F:\\data_zf\\changeTcasthtest.txt");
		if (!fout)
			cout << "changeTcasthtest is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				for (int j = 0; j < Section; j++)
					fout << htest[i][j] << ",";
				fout << endl;
			}
		}
		fout.close();

		fout.open("F:\\data_zf\\changeTcastfitnesstest.txt");
		if (!fout)
			cout << "changeTcastfitnesstest is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				fout << fitness[i] << ",";

				fout << endl;
			}
		}
		fout.close();

		fout.open("F:\\data_zf\\changeTcastgfitnesstest.txt");
		if (!fout)
			cout << "changeTcastgfitnesstest is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				fout << gfitness[i] << ",";

				fout << endl;
			}
		}
		fout.close();

		fout.open("F:\\data_zf\\changeTcastc.txt");
		if (!fout)
			cout << "changeTcastc is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				fout << c[i] << ",";

				fout << endl;
			}
		}							
		fout.close();

		fout.open("F:\\data_zf\\changeTcaststep.txt");
		if (!fout)
			cout << "changeTcaststep is not open" << endl;
		else
		{
			for (int i = 0; i < TestIter; i++)
			{
				fout << testArray[i] << ",";
				fout << endl;
			}
		}		
		fout.close();

	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.


Error:
	hipFree(T_Init);
	hipFree(dev_T_New);
	hipFree(dev_T_Last);
	hipFree(dev_ccml);
	hipFree(dev_H_Init);
	hipFree(JacobianMatrix);
	hipFree(JacobianG0);
	hipFree(JacobianG1);
	hipFree(JacobianG2);
	hipFree(JacobinTZgradient);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.

__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd)
{
	float Ts = 1462.0, Tl = 1518.0, lamds = 30, lamdl = 50, phos = 7000, phol = 7500, ce = 540.0, L = 265600.0, fs = 0.0;
	if (T<Ts)
	{
		fs = 0;
		*pho = phos;
		*lamd = lamds;
		*Ce = ce;
	}

	if (T >= Ts&&T <= Tl)
	{
		fs = (T - Ts) / (Tl - Ts);
		*pho = fs*phos + (1 - fs)*phol;
		*lamd = fs*lamds + (1 - fs)*lamdl;
		*Ce = ce + L / (Tl - Ts);
	}

	if (T>Tl)
	{
		fs = 1;
		*pho = phol;
		*lamd = lamdl;
		*Ce = ce;
	}

}

__device__ float Boundary_Condition(int j, float dy, float *ccml_zone, float *H_Init)
{
	float YLabel, h = 0.0;
	YLabel = j*dy;

	for (int i = 0; i < Section; i++)
	{
		if (YLabel >= *(ccml_zone + i) && YLabel <= *(ccml_zone + i + 1))
			h = *(H_Init + i);
	}
	return h;
}

float* Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T, float num)
{
	float y;
	int count = 0;
	int i = 0;

	float* Mean_Temperature;
	Mean_Temperature = new float[Section];
	for (int i = 0; i < Section; i++)
	{
		Mean_Temperature[i] = 0.0;
		for (int j = 0; j < ny - num; j++)
		{
			y = j * dy;
			if (y > *(ccml + i) && y <= *(ccml + i + 1))
			{
				Mean_Temperature[i] = Mean_Temperature[i] + T[nx * nz * j + (int)(num * nz) + int((nx - 1) / 2)];
				count++;
			}
		}
		Mean_Temperature[i] = Mean_Temperature[i] / float(count);
		count = 0;
	}
	return Mean_Temperature;
}
float stop_criterion() {
	float norm_g = 0.0;
	for (int i = 0; i <= N - 1; i++)
		norm_g = norm_g + g[i] * g[i];
	norm_g = sqrt(norm_g);
	//printf("norm_g=%f\n", norm_g);
	return(norm_g);
}

float update_c(float norm_g[], float c0,int iter) {//����Luh�����й�ʽ20
	float dM = 2.0, r = 0.5, p = 0.0, c = 10;
	if (iter > 0)
	{
		p = 1.0 - 1.0 / pow(iter, r);//p�ĸ��¹�ʽ67

		alfa[iter - 1] = 1.0 - 1.0 / (dM*pow(iter, p));//���Ӧ����iter��������67
	}

	if (iter <= StaticIter)
		return c0;

	else
	{		
		/*for(int i=0;i<iter;i++)
		printf("  alfa=%f",alfa[i]);
		printf("\n");*/

		c = c0*norm_g[0] / norm_g[iter - 1];//c�ĵ�����ʽһ����20

		//printf("norm_g[0]=%f\n", norm_g[0]);
		//printf("norm_g[iter-1]=%f\n", norm_g[iter-1]);

		for (int i = 0; i <= iter - 1; i++)
			c = c*alfa[i];//c�ĵ�����ʽ������20
	}

	return(c);
}
float *relationshiphandQ(float *h_Init, float* Q_air)
{
	float hx[CoolSection] = { 56.5,40.2,40.2,40.2,40.2,40.2,40.2,40.2 };
	float rw[CoolSection] = { 0.845,0.568,0.568,0.568,0.568,0.568,0.568,0.568 };
	float ra[CoolSection] = { 0.2,0.1902,0.1902,0.1902,0.1902,0.1902,0.1902,0.1902 };
	float hr[CoolSection] = { 0.15,0.082,0.082,0.082,0.082,0.082,0.082,0.082 };
	float Sw[CoolSection] = { 1.8,3.86,1.8,1.8, 1.8, 1.8, 1.8, 1.8 };
	float Sl[CoolSection] = { 0.5,0.8,2.5,1.8,4.0,3.5,6.0,8.9 };

	float *Q_water;
	Q_water = new float[CoolSection];
	for (int i = 0; i < CoolSection; i++)
	{
		Q_water[i] = pow((h_Init[i+MoldSection] - hr[i]) / hx[i] / pow(Q_air[i] / (Sl[i] * Sw[i]), ra[i]), 1 / rw[i])*(Sl[i] * Sw[i]);
	}
	return Q_water;
}
float *calculateThickness(float *T_result, int nx, int ny, int nz, float dy, float *ccml,float Ts,float thick) {
	float y;
	int count = 0;
	float *Mean_Temperature;
	Mean_Temperature = new float[Section];
	int *thickness;
	thickness = new int[Section];
	float *res_thickness;
	res_thickness = new float[Section];
	for (int i = 0; i < Section; i++)	
	{
		for (thickness[i] = 0; thickness[i] <nx; thickness[i]++){
			Mean_Temperature[i] = 0.0;
			for (int j = 0; j < ny; j++)
			{
				y = j * dy;
				if (y > *(ccml + i) && y <= *(ccml + i + 1))
				{
					Mean_Temperature[i] = Mean_Temperature[i] + T_result[nx * nz * j + thickness[i] * nz + int((nx - 1) / 2)];
					count++;
				}
			}
			Mean_Temperature[i] = Mean_Temperature[i] / float(count);
			count = 0;
			if (Mean_Temperature[i] > Ts)
				break;
	   }	
		res_thickness[i]=thickness[i] * thick / nx;
	}
	return res_thickness;
}
