#include "hip/hip_runtime.h"
//3D_shared_memory
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "book.h"
#include "gridcheck.h"

# define Section 12  // number of cooling sections
# define num_threadsx 3
# define num_threadsy 15
# define num_threadsz 21
# define num_blocksx 7
# define num_blocksy 200
# define num_blocksz 1
hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, float *, float *);
__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd);
__device__ float Boundary_Condition(int j, float dx, float *ccml_zone, float *H_Init);

__global__ void addKernel(float *T_New, float *T_Last, float *ccml, float *H_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, bool disout)
{
	//����shared memory 
	const int M = num_threadsx, N = num_threadsy, Z = num_threadsz;//����shared memory ��С
	__shared__ float shared[M+2][N+2][Z+2];//ֻ��һά�Ƕ�̬�ģ��������ڵ��õ�ʱ��������С����ά�ľ�ֱ�Ӷ����С

	int tidx = threadIdx.x;
	int posx = threadIdx.x + 1;
	int tidy = threadIdx.y;
	int posy = threadIdx.y+1;
	int tidz = threadIdx.z;
	int posz = threadIdx.z + 1;
	int i = blockIdx.x*blockDim.x+tidx;
	int j = blockIdx.y*blockDim.y + tidy;
	int m = blockIdx.z*blockDim.z + tidz;
	int idx = j * nx * nz + m * nx + i;
	int ND = nx * nz;
	int D = nx;

	float pho, Ce, lamd; // physical parameters pho represents desity, Ce is specific heat and lamd is thermal conductivity
	float a, T_Up, T_Down, T_Right, T_Left, T_Forw, T_Back, h = 100.0, Tw = 30.0, Vcast = -0.02, T_Cast = 1558.0;


	//�ж�����
	int schedule;
	if (j == 0) //1��
		schedule = 1;
	if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10��
		schedule = 2;
	if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11��
		//schedule = 2;
		schedule = 3;
    if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12��
		//schedule = 2;
		schedule = 3;
	if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13��
		schedule = 3;
		//schedule = 2;
	if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14��
		//schedule = 2;
		schedule = 3;
	if (j == (ny - 1) && i == 0 && m == 0)  //15��
		//schedule = 2;
		schedule = 3;
	if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16��
		//schedule = 2;
		schedule = 3;
	if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17��
		//schedule = 2;
		schedule = 3;
	if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18��
		//schedule = 2;
		schedule = 3;
	if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19��
		schedule = 3;
	if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20��
		schedule = 4;
	if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21��
		schedule = 3;
	if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22��
		schedule = 3;
	if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23��
		//schedule = 4;
		schedule = 3;
	if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24��
		//schedule = 4;
    	schedule = 3;
	if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25��
		schedule = 5;
	if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26��
		schedule = 6;
	if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1))//27�ڲ�
		schedule = 0;
	//��ֵ���
	shared[posx][posy][posz] = T_Last[idx];//��shared ��ά��������ʾ��ά�¶ȳ�

	if (blockIdx.x > 0) {
		if(threadIdx.x==0) shared[0][posy][posz] = T_Last[j*ND+m*D+i-1];
	}//��֤����blockx����߽��¶�
	if (blockIdx.x < gridDim.x-1) {
		if (threadIdx.x == 0) shared[M + 1][posy][posz] = T_Last[j*ND + m*D +i+ M + 1];
	}//��֤����blockx���ұ߽��¶�
	if (blockIdx.y > 0) {
		if (threadIdx.y == 0) shared[posx][0][posz] = T_Last[(j-1)*ND + m*D +i];
	}//��֤����blockY����߽��¶�
	if (blockIdx.x < gridDim.y - 1) {
		if (threadIdx.y == 0) shared[posx][N+1][posz] = T_Last[(j+N+1)*ND + m*D + i];
	}//��֤����blockY���ұ߽��¶�
	if (blockIdx.z > 0) {
		if (threadIdx.z == 0) shared[posx][posy][0] = T_Last[j*ND + (m-1)*D +i];
	}//��֤����blockx����߽��¶�
	if (blockIdx.x < gridDim.z - 1) {
		if (threadIdx.z == 0) shared[posx][posy][Z+1] = T_Last[j*ND + (m+Z+1)*D  + i];
	}//��֤����blockx���ұ߽��¶�
	__syncthreads();

	Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
	a = (lamd) / (pho*Ce);
	//h = Boundary_Condition(j, dy, ccml, H_Init);
	switch (schedule)
	{
	case 0:
		//T_New[idx] = T_Cast;
		T_Up = shared[posx+1][posy][posz];// T_Last[idx + 1];
		T_Down = shared[posx-1][posy][posz];// T_Last[idx - 1];
		T_Right = shared[posx][posy+1][posz];// T_Last[idx + ND];
		T_Left = shared[posx][posy-1][posz]; //T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz+1]; //T_Last[idx + D];
		T_Back = shared[posx][posy][posz-1]; //T_Last[idx - D];
		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
			break;
	case 1:
		T_New[idx] = T_Cast;
		break;
	case 2:
		//T_New[idx] = T_Cast;

		T_Up = shared[posx+1][posy][posz];// T_Last[idx + 1];
		T_Down = shared[posx-1][posy][posz];//  T_Last[idx - 1];
		T_Right = shared[posx][posy-1][posz];//  T_Last[idx - ND];
		T_Left = shared[posx][posy-1][posz];// T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz+1];// T_Last[idx + D];
		T_Back = shared[posx][posy][posz-1];// T_Last[idx - D];
		T_New[idx] = (a*tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ (a*tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
			break;
	case 3:
		//T_New[idx] = T_Cast;		
		h = Boundary_Condition(j, dy, ccml, H_Init);
		T_Up = shared[posx+1][posy][posz];//T_Last[idx + 1];
		T_Down = shared[posx-1][posy][posz];//T_Last[idx - 1];
		T_Right = shared[posx][posy+1][posz];// T_Last[idx + ND];
		T_Left = shared[posx][posy-1][posz];//T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz+1];//T_Last[idx + D];
		T_Back = shared[posx][posy][posz+1] - 2 * dz * h * (shared[posx][posy][posz] - Tw) / lamd;// T_Last[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		break;
	case 4:
		//T_New[idx] = T_Cast;		
		T_Up = shared[posx+1][posy][posz];//T_Last[idx + 1];
		T_Down = shared[posx+1][posy][posz];// T_Last[idx + 1];
		T_Right = shared[posx][posy+1][posz];//T_Last[idx + ND];
		T_Left = shared[posx][posy-1][posz];//T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz-1];//T_Last[idx - D];
		T_Back = shared[posx][posy][posz-1];//T_Last[idx - D];
		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		break;
	case 5:
		//T_New[idx] = T_Cast;
		h = Boundary_Condition(j, dy, ccml, H_Init);
		T_Up = shared[posx+1][posy][posz];// T_Last[idx + 1];
		T_Down = shared[posx+1][posy][posz] - 2 * dx * h * (shared[posx][posy][posz] - Tw) / lamd;//T_Last[idx + 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
		T_Right = shared[posx][posy+1][posz];// T_Last[idx + ND];
		T_Left = shared[posx][posy-1][posz];// T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz+1];//T_Last[idx + D];
		T_Back = shared[posx][posy][posz-1];// T_Last[idx - D];
		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		break;
	case 6:
		//T_New[idx] = T_Cast;
		h = Boundary_Condition(j, dy, ccml, H_Init);
		T_Up = shared[posx-1][posy][posz] - 2 * dx * h * (shared[posx][posy][posz] - Tw) / lamd;// T_Last[idx - 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
		T_Down = shared[posx-1][posy][posz];// T_Last[idx - 1];
		T_Right = shared[posx][posy+1][posz];// T_Last[idx + ND];
		T_Left = shared[posx][posy-1][posz];// T_Last[idx - ND];
		T_Forw = shared[posx][posy][posz+1];// T_Last[idx + D];
		T_Back = shared[posx][posy][posz-1];// T_Last[idx - D];
		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*shared[posx][posy][posz]
			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		break;
	}
	T_Last[idx] = T_New[idx];
	//if (disout) {
	//	Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
	//	a = (lamd) / (pho*Ce);
	//	h = Boundary_Condition(j, dy, ccml, H_Init);
	//	if (j == 0) //1
	//	{
	//		T_New[idx] = T_Cast;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = (a*tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ (a*tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx - 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m == 0)  //15
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx - 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
	//	{
	//		//T_New[idx] = 1550.0;
	//		T_Up = T_Last[idx - 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx - ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx - 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx + D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx - 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx - D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx + 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx - 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else  //27
	//	{
	//		//T_New[idx] = T_Cast;
	//		T_Up = T_Last[idx + 1];
	//		T_Down = T_Last[idx - 1];
	//		T_Right = T_Last[idx + ND];
	//		T_Left = T_Last[idx - ND];
	//		T_Forw = T_Last[idx + D];
	//		T_Back = T_Last[idx - D];
	//		T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}
	//}

	//else
	//{
	//	Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
	//	a = (lamd) / (pho*Ce);
	//	h = Boundary_Condition(j, dy, ccml, H_Init);
	//	if (j == 0) //1
	//	{
	//		T_Last[idx] = T_Cast;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx - 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m == 0)  //15
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx - 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
	//	{
	//		//T_Last[idx] = 1550.0;
	//		T_Up = T_New[idx - 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx - ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx - 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx + D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx - 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx - D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx + 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx - 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}

	//	else  //27
	//	{
	//		//T_Last[idx] = T_Cast;
	//		T_Up = T_New[idx + 1];
	//		T_Down = T_New[idx - 1];
	//		T_Right = T_New[idx + ND];
	//		T_Left = T_New[idx - ND];
	//		T_Forw = T_New[idx + D];
	//		T_Back = T_New[idx - D];
	//		T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
	//			+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
	//	}
	//}
}

int main()
{
	const int nx = 21, ny = 3000, nz = 21;   // nx is the number of grid in x direction, ny is the number of grid in y direction.
	//int num_blocksx = 1, num_blocksy = 1, num_blocksz = 1, num_threadsx = 1, num_threadsy = 1, num_threadsz = 1;// block number(1D)  thread number in x and y dimension(2D)
	int tnpts = 10000;  // time step
	float T_Cast = 1558.0, Lx = 0.25, Ly = 28.599, Lz = 0.25, t_final = 2000.0, dx, dy, dz, tao;  // T_Cast is the casting temperature Lx and Ly is the thick and length of steel billets
	float *T_Init;
	float ccml[Section + 1] = { 0.0,0.2,0.4,0.6,0.8,1.0925,2.27,4.29,5.831,9.6065,13.6090,19.87014,28.599 }; // The cooling sections
	float H_Init[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections

	T_Init = (float *)calloc(nx * ny * nz, sizeof(float));  // Initial condition

	/*num_threadsx = 7;
	num_threadsy = 20;
	num_threadsz = 7;
	num_blocksx = nx/num_blocksx;
	num_blocksy = nx / num_blocksy;
	num_blocksz = nx / num_blocksz;*/

	for (int m = 0; m < nz; m++)
		for (int j = 0; j < ny; j++)
	       for (int i = 0; i < nx; i++)
			   T_Init[nx * ny * m + j * nx + i] = T_Cast;  // give the initial condition

	dx = Lx / (nx - 1);            // the grid size x
	dy = Ly / (ny - 1);            // the grid size y
	dz = Lz / (nz - 1);            // the grid size y
	tao = t_final / (tnpts - 1);   // the time step size
	//gridcheck(dx, dy, tao);

	printf("Casting Temperature = %f ", T_Cast);
	printf("\n");
	printf("The thick of steel billets(m) = %f ", Lx);
	printf("\n");
	printf("The length of steel billets(m) = %f ", Ly);
	printf("\n");
	printf("The length of steel billets(m) = %f ", Lz);
	printf("\n");
	printf("dx(m) = %f ", dx);
	printf("dy(m) = %f ", dy);
	printf("dz(m) = %f ", dz);
	printf("tao(s) = %f ", tao);
	printf("\n");
	printf("simulation time(s) = %f\n ", t_final);

	clock_t timestart = clock();
	hipError_t cudaStatus = addWithCuda(T_Init, dx, dy, dz, tao, nx, ny, nz, tnpts, ccml, H_Init );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	clock_t timeend = clock();

	printf("running time = %d(millisecond)", (timeend - timestart));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, float *ccml, float *H_Init)
{
	float *dev_T_New, *dev_T_Last, *dev_ccml, *dev_H_Init; // the point on GPU
	float *T_Result;
	const int Num_Iter = 2000;                         // The result can be obtained by every Num_Iter time step
	volatile bool dstOut = true;
	FILE *fp = NULL;

	T_Result = (float *)calloc(nx * ny * nz, sizeof(float)); // The temperature of steel billets

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_ccml, (Section + 1) * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init, Section * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_T_Last, T_Init, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_ccml, ccml, (Section + 1) * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init, Section * sizeof(float), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(num_threadsx, num_threadsy,num_threadsz);
	dim3 BlocksPerGrid(num_blocksx,num_blocksy,num_blocksz);
	// Launch a kernel on the GPU with one thread for each element.
	for (int i = 0; i < tnpts; i++)
	{
		addKernel << <BlocksPerGrid, threadsPerBlock >> >(dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut);
		dstOut = !dstOut;

		if (i % Num_Iter == 0) {
			HANDLE_ERROR(hipMemcpy(T_Result, dev_T_Last, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
			printf("time_step = %d\n  simulation time is %f\n", i, i*tao);
			printf("%f, %f, %f, %f", T_Result[0], T_Result[(nx - 1)*(ny - 1)*(nz - 1) - nx], T_Result[(nx - 1)*(ny - 1)*(nz - 1) - nx * nz],T_Result[(nx - 1)*(ny - 1)*(nz - 1)]);
			printf("\n");
		}
	}

	fp = fopen("F:\\Temperature3DGPU_shared_memmory_Static.txt", "w");
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			for (int m = 0; m < nz; m++)
				fprintf(fp, " %f", T_Result[ nx * nz * j  + i * nz + m]);
			fprintf(fp, "\n");
		}
		fprintf(fp, "\n");
	}
	fclose(fp);

	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.


Error:
	hipFree(dev_T_New);
	hipFree(dev_T_Last);
	hipFree(dev_ccml);
	hipFree(dev_H_Init);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.

__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd)
{
	float Ts = 1462.0, Tl = 1518.0, lamds = 30, lamdl = 50, phos = 7000, phol = 7500, ce = 540.0, L = 265600.0, fs = 0.0;
	if (T<Ts)
	{
		fs = 0;
		*pho = phos;
		*lamd = lamds;
		*Ce = ce;
	}

	if (T >= Ts&&T <= Tl)
	{
		fs = (T - Ts) / (Tl - Ts);
		*pho = fs*phos + (1 - fs)*phol;
		*lamd = fs*lamds + (1 - fs)*lamdl;
		*Ce = ce + L / (Tl - Ts);
	}

	if (T>Tl)
	{
		fs = 1;
		*pho = phol;
		*lamd = lamdl;
		*Ce = ce;
	}

}

__device__ float Boundary_Condition(int j, float dy, float *ccml_zone, float *H_Init)
{
	float YLabel, h = 0.0;
	YLabel = j*dy;

	for (int i = 0; i < Section; i++)
	{
		if (YLabel >= *(ccml_zone + i) && YLabel <= *(ccml_zone + i + 1))
			h = *(H_Init + i);
	}
	return h;
}